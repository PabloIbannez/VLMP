#include "hip/hip_runtime.h"
#include "UAMMDstructured.cuh"

using namespace uammd::structured;

int main(int argc, char *argv[]) {

    if (argc < 2) {
        uammd::System::log<uammd::System::CRITICAL>("No input file provided!");
        return EXIT_FAILURE;
    }

    startSelfStartingSimulation(argv[1]);

    return EXIT_SUCCESS;
}
